#include "hip/hip_runtime.h"
#include <redistance.h>
#include <redistance_kernels.h>
#include <Vec.h>
#include <math.h>
#include <stdio.h>
#include <cutil.h>

#include "cusp/print.h"

void redistance::ReInitTsign(TriMesh* mesh, Vector_d& vertT_after_permute_d, int nparts, int largest_vert_part, int largest_ele_part, int largest_num_inside_mem, int full_num_ele,
    Vector_d& vert_after_permute_d, IdxVector_d& vert_offsets_d,
    IdxVector_d& ele_after_permute_d, IdxVector_d& ele_offsets_d, Vector_d& ele_local_coords_d, IdxVector_d& mem_location_offsets, IdxVector_d& mem_locations,
    IdxVector_d& part_label_d, IdxVector_d& block_xadj, IdxVector_d& block_adjncy)
{
  int nn = mesh->vertices.size();
  int nthreads = 256;
  int nblocks = min((int)ceil((double)nn / nthreads), 65535);
  cudaSafeCall((kernel_reinit_Tsign << <nblocks, nthreads >> >(nn, CAST(vertT_after_permute_d), CAST(m_Tsign_d))));
}

void redistance::FindSeedPoint(const IdxVector_d& old_narrowband, const int num_old_narrowband, TriMesh* mesh, Vector_d& vertT_after_permute_d, int nparts,
    int largest_vert_part, int largest_ele_part, int largest_num_inside_mem, int full_num_ele,
    Vector_d& vert_after_permute_d, IdxVector_d& vert_offsets_d,
    IdxVector_d& ele_after_permute_d, IdxVector_d& ele_offsets_d, Vector_d& ele_local_coords_d,
    IdxVector_d& mem_location_offsets, IdxVector_d& mem_locations,
    IdxVector_d& part_label_d, IdxVector_d& block_xadj, IdxVector_d& block_adjncy)
{
  int ne = mesh->faces.size();
  int nn = mesh->vertices.size();
  int nnb = num_old_narrowband;
  thrust::fill(m_DT_d.begin(), m_DT_d.end(), LARGENUM);
  m_active_block_list_d[0] = 0;
  if (nnb == 0)
  {
    thrust::fill(m_Label_d.begin(), m_Label_d.end(), FarPoint);
    int nthreads = largest_ele_part;
    int nblocks = nparts;

    cudaSafeCall((kernel_seedlabel << <nblocks, nthreads >> >(nn, full_num_ele,
            CAST(vert_after_permute_d),
            CAST(vert_offsets_d),
            CAST(ele_after_permute_d),
            CAST(ele_offsets_d),
            CAST(m_Label_d),
            CAST(vertT_after_permute_d),
            CAST(m_DT_d),
            CAST(m_active_block_list_d))));
  }
  else
  {
    thrust::fill(m_Label_d.begin(), m_Label_d.end(), FarPoint);
    int nthreads = largest_ele_part;
    int nblocks = nnb;
    cudaSafeCall((kernel_seedlabel_narrowband << <nblocks, nthreads >> >(nn, full_num_ele, CAST(old_narrowband),
            CAST(vert_after_permute_d),
            CAST(vert_offsets_d),
            CAST(ele_after_permute_d),
            CAST(ele_offsets_d),
            CAST(m_Label_d),
            CAST(vertT_after_permute_d),
            CAST(m_DT_d),
            CAST(m_active_block_list_d))));

  }
}

void redistance::GenerateData(IdxVector_d& new_narrowband, int& new_num_narrowband, double bandwidth, int stepcount, TriMesh* mesh, Vector_d& vertT_after_permute_d,
    int nparts, int largest_vert_part, int largest_ele_part, int largest_num_inside_mem, int full_num_ele,
    Vector_d& vert_after_permute_d, IdxVector_d& vert_offsets_d,
    IdxVector_d& ele_after_permute_d, IdxVector_d& ele_offsets_d, Vector_d& ele_local_coords_d,
    IdxVector_d& mem_location_offsets, IdxVector_d& mem_locations,
    IdxVector_d& part_label_d, IdxVector_d& block_xadj, IdxVector_d& block_adjncy, bool verbose)
{
  int nn = mesh->vertices.size();
  int totalIterationNumber = 0;
  int nblocks, nthreads, shared_size;
  int NUM_ITER = 10;
  int nTotalIter = 0;
  int numActive = m_active_block_list_d[0];
  thrust::copy(m_DT_d.begin(), m_DT_d.end(), DT_d_out.begin());
  thrust::fill(d_vert_con.begin(), d_vert_con.end(), 0);
  thrust::fill(d_block_con.begin(), d_block_con.end(), 0);

  thrust::copy(m_active_block_list_d.begin() + 1, m_active_block_list_d.begin() + 1 + numActive, h_ActiveList.begin());
  h_BlockLabel.assign(nparts, FarPoint);
  
  
  while(numActive > 0)
  {
    if (verbose) {
      size_t act = numActive / 3;
      for (size_t ab = 0; ab < 60; ab++) {
        if (ab < act)
          printf("=");
        else
          printf(" ");
      }
      printf(" %d Active blocks.\n", numActive);
    }
    //    printf("nTotalIter = %d, numActive=%d\n", nTotalIter, numActive);
	
	
    //////////////////////////////////////////////////////////////////
    // 1.update the values of nodes in triangles/tets with local _solver
	/////////////////////////////////////////////////////////////////
	nTotalIter++;
    totalIterationNumber += numActive;

    nblocks = numActive;
    nthreads = largest_ele_part;
    m_active_block_list_d = h_ActiveList;
    shared_size = sizeof(double)* 3 * largest_ele_part + sizeof(short)*largest_vert_part*largest_num_inside_mem;
    cudaSafeCall((kernel_update_values << <nblocks, nthreads, shared_size >> >(CAST(m_active_block_list_d), CAST(m_Label_d), largest_ele_part, largest_vert_part, full_num_ele,
            CAST(ele_after_permute_d), CAST(ele_offsets_d),
            CAST(vert_offsets_d), CAST(m_DT_d),
            CAST(ele_local_coords_d), largest_num_inside_mem, CAST(mem_locations), CAST(mem_location_offsets),
            NUM_ITER, CAST(DT_d_out), CAST(d_vert_con))));
    nthreads = largest_vert_part;
    cudaSafeCall((CopyOutBack << <nblocks, nthreads >> >(CAST(m_active_block_list_d),
            CAST(vert_offsets_d), CAST(m_DT_d), CAST(DT_d_out))));

    //////////////////////////////////////////////////////////////////
    // check if tri/tet is converged with reduction operation
	// compute the minimum value of the all the nodes in tri/tet
	/////////////////////////////////////////////////////////////////
    if(nthreads <= 32)
    {
      cudaSafeCall((run_reduction_bandwidth < 32 > << <nblocks, 32 >> > (CAST(d_vert_con), CAST(d_block_con), CAST(m_active_block_list_d),
              CAST(DT_d_out), CAST(d_block_vertT_min), CAST(vert_offsets_d))));
    }
    else if(nthreads <= 64)
    {
      cudaSafeCall((run_reduction_bandwidth < 64 > << <nblocks, 64 >> > (CAST(d_vert_con), CAST(d_block_con), CAST(m_active_block_list_d),
              CAST(DT_d_out), CAST(d_block_vertT_min), CAST(vert_offsets_d))));
    }
    else if(nthreads <= 128)
    {
      cudaSafeCall((run_reduction_bandwidth < 128 > << <nblocks, 128 >> > (CAST(d_vert_con), CAST(d_block_con), CAST(m_active_block_list_d),
              CAST(DT_d_out), CAST(d_block_vertT_min), CAST(vert_offsets_d))));
    }
    else if(nthreads <= 256)
    {
      cudaSafeCall((run_reduction_bandwidth < 256 > << <nblocks, 256 >> > (CAST(d_vert_con), CAST(d_block_con), CAST(m_active_block_list_d),
              CAST(DT_d_out), CAST(d_block_vertT_min), CAST(vert_offsets_d))));
    }
    else if(nthreads <= 512)
    {
      cudaSafeCall((run_reduction_bandwidth < 512 > << <nblocks, 512 >> > (CAST(d_vert_con), CAST(d_block_con), CAST(m_active_block_list_d),
              CAST(DT_d_out), CAST(d_block_vertT_min), CAST(vert_offsets_d))));
    }
    else
    {
      printf("Error: nthreads greater than 256!!!\n");
    }
    thrust::copy(d_block_con.begin(), d_block_con.end(), h_block_con.begin());
    h_block_vertT_min = d_block_vertT_min;
	
	/////////////////////////
	// if a is converged && phi min < w then
	// add neighboring patches of a into a temporary list
	// clear active list L
	/////////////////////////
    int nOldActiveBlock = numActive;
    numActive = 0;
    h_ActiveListNew.clear();
    for(int i = 0; i < nOldActiveBlock; i++)
    {
      int currBlkIdx = h_ActiveList[i];
      h_BlockLabel[currBlkIdx] = FarPoint;
      if(!h_block_con[currBlkIdx]) // if not converged
      {
        h_BlockLabel[currBlkIdx] = ActivePoint;
      }
    }
    for(int i = 0; i < nOldActiveBlock; i++)
    {
      int currBlkIdx = h_ActiveList[i];

      if(h_block_con[currBlkIdx] && h_block_vertT_min[currBlkIdx] < bandwidth) //converged
      {
        int start = block_xadj_h[currBlkIdx];
        int end = block_xadj_h[currBlkIdx + 1];

        for(int iter = 0; iter < end - start; iter++)
        {
          int currIdx = block_adjncy_h[iter + start];
          if(h_BlockLabel[currIdx] == FarPoint)
          {
            h_BlockLabel[currIdx] = ActivePoint;
            h_ActiveListNew.push_back(currIdx);
          }
        }
      }
    }

    for(int i = 0; i < nOldActiveBlock; i++)
    {
      int currBlkIdx = h_ActiveList[i];
      if(!h_block_con[currBlkIdx]) // if not converged
      {
        h_ActiveList[numActive++] = currBlkIdx;
      }
    }
	
    //////////////////////////////////////////////////////////////////
    // run solver only once for neighbor blocks of converged block
    // current active list contains active blocks and neighbor blocks of
    // any converged blocks
	//////////////////////////////////////////////////////////

    if(h_ActiveListNew.size() > 0)
    {
      int numActiveNew = h_ActiveListNew.size();
      m_active_block_list_d = h_ActiveListNew;
      nblocks = numActiveNew;
      nthreads = largest_ele_part;

      int sharedSize = sizeof(double)* 3 * largest_ele_part + 
        sizeof(short)*largest_vert_part*largest_num_inside_mem;
      ////DEBUG find bad memory
      /*
      IdxVector_h active_block_list_h(m_active_block_list_d);
      std::vector<int> act_blk_lst;
      for (size_t i = 0; i < active_block_list_h.size(); i++) {
        act_blk_lst.push_back(active_block_list_h[i]);
      }
      IdxVector_h labels_h(m_Label_d);
      std::vector<int> labels;
      for (size_t i = 0; i < labels_h.size(); i++) {
        labels.push_back(labels_h[i]);
      }
      IdxVector_h ele_afterPermute_h(ele_after_permute_d);
      std::vector<int> ele_after_permute;
      for (size_t i = 0; i < ele_afterPermute_h.size(); i++) {
        ele_after_permute.push_back(ele_afterPermute_h[i]);
      }
      IdxVector_h eleOffsets_h(ele_offsets_d);
      std::vector<int> eleOffsets;
      for (size_t i = 0; i < eleOffsets_h.size(); i++) {
        eleOffsets.push_back(eleOffsets_h[i]);
      }
      IdxVector_h vertOffsets_h(vert_offsets_d);
      std::vector<int> vertOffsets;
      for (size_t i = 0; i < vertOffsets_h.size(); i++) {
        vertOffsets.push_back(vertOffsets_h[i]);
      }*/


      cudaSafeCall((kernel_run_check_neghbor << <nblocks, nthreads, shared_size >> >(
        CAST(m_active_block_list_d),
        CAST(m_Label_d),
        largest_ele_part,
        largest_vert_part,
        full_num_ele,
        CAST(ele_after_permute_d),
        CAST(ele_offsets_d),
        CAST(vert_offsets_d), //
        CAST(m_DT_d),
        CAST(ele_local_coords_d), 
        largest_num_inside_mem, 
        CAST(mem_locations), 
        CAST(mem_location_offsets), 1,
        CAST(DT_d_out), 
        CAST(d_vert_con))));

      if (sharedSize <= 0) {
        printf("Error: zero shared size");
      }


      ////////////////////////////////////////////////////////////////
      // check if a is converged with reduction operation
      ////////////////////////////////////////////////////////////////
      nthreads = largest_vert_part;
      run_reduction << <nblocks, nthreads >> >(
        CAST(d_vert_con), 
        CAST(d_block_con),
        CAST(m_active_block_list_d),
        CAST(vert_offsets_d));

      //////////////////////////////////////////////////////////////////
      // update active list
      // read back active volume from the device and add
      // active block to active list on the host memory
	  //////////////////////////////////////////////////////////////////
      h_block_con = d_block_con;
      for(int i = 0; i < h_ActiveListNew.size(); i++)
      {
        int currBlkIdx = h_ActiveListNew[i];
        if(!h_block_con[currBlkIdx]) // false : activate block (not converged)
        {
          h_ActiveList[numActive++] = currBlkIdx;
        }
        else h_BlockLabel[currBlkIdx] = FarPoint;
      }
    }
  }
  
  //////////////////////////////////
  //compute new narrow band list
  ///////////////////////////////////////
  nblocks = nparts;
  nthreads = largest_vert_part;
  tmp_new_narrowband[0] = 0;

  if(nthreads <= 32)
  {
    cudaSafeCall((kernel_compute_new_narrowband < 32 > << <nblocks, 32 >> > (CAST(tmp_new_narrowband), CAST(m_DT_d), CAST(vert_offsets_d), bandwidth)));
  }
  else if(nthreads <= 64)
  {
    cudaSafeCall((kernel_compute_new_narrowband < 64 > << <nblocks, 64 >> >(CAST(tmp_new_narrowband), CAST(m_DT_d), CAST(vert_offsets_d), bandwidth)));
  }
  else if(nthreads <= 128)
  {
    cudaSafeCall((kernel_compute_new_narrowband < 128 > << <nblocks, 128 >> >(CAST(tmp_new_narrowband), CAST(m_DT_d), CAST(vert_offsets_d), bandwidth)));
  }
  else if(nthreads <= 256)
  {
    cudaSafeCall((kernel_compute_new_narrowband < 256 > << <nblocks, 256 >> >(CAST(tmp_new_narrowband), CAST(m_DT_d), CAST(vert_offsets_d), bandwidth)));
  }
  else if(nthreads <= 512)
  {
    cudaSafeCall((kernel_compute_new_narrowband < 512 > << <nblocks, 512 >> >(CAST(tmp_new_narrowband), CAST(m_DT_d), CAST(vert_offsets_d), bandwidth)));
  }
  else
  {
    printf("Error: nthreads greater then 256!!!\n");
  }
  int numb = tmp_new_narrowband[0];
  new_num_narrowband = numb;
  nblocks = numb;
  thrust::copy(m_DT_d.begin(), m_DT_d.end(), vertT_after_permute_d.begin());

  thrust::copy(tmp_new_narrowband.begin() + 1, tmp_new_narrowband.begin() + numb + 1, new_narrowband.begin());
  nthreads = 256;
  nblocks = min((int)ceil((double)nn / nthreads), 65535);
  cudaSafeCall((kernel_recover_Tsign_whole << <nblocks, nthreads >> >(nn, CAST(vertT_after_permute_d), CAST(m_Tsign_d))));
}
