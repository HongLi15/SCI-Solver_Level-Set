#include "hip/hip_runtime.h"
#include <meshFIM2d.h>
#include <Vec.h>
#include <cmath>
#include <cstdio>
#include <cstdlib>
#include <meshFIM2d_kernels.h>
#include <cutil.h>
#include <cusp/detail/format_utils.h>
#include <cusp/print.h>
#include <thrust/functional.h>
#include <sstream>

extern "C"
{
#include <metis.h>
}

void meshFIM2d::writeFLD()
{
  size_t nv = m_meshPtr->vertices.size();
  size_t nt = m_meshPtr->faces.size();
  FILE* matfile;
  FILE* fldfile;
  matfile = fopen("result.txt", "w+");
  fldfile = fopen("result.fld", "w+");
  fprintf(fldfile, "SCI\nASC\n2\n{@1 {GenericField<TriSurfMesh<TriLinearLgn<Point>>,ConstantBasis<float>,vector<float>> 3 {Field 3 {PropertyManager 2 1 \"name\" @2 {Property<string> 2 0 \"TorsoVolumeMesh\" }\n}\n}\n{@3 {TriSurfMesh<TriLinearLgn<Point>> 4 {Mesh 2 {PropertyManager 2 0 }\n}\n");
  fprintf(fldfile, "{STLVector 2 %d ", nv);
  for (size_t i = 0; i < nv; i++)
  {
    fprintf(fldfile, "{%.12f %.12f %.12f}", m_meshPtr->vertices[i][0], m_meshPtr->vertices[i][1], m_meshPtr->vertices[i][2]);
  }
  fprintf(fldfile, "}\n{STLIndexVector 1 %d 8 ", nt * 3);
  for (size_t i = 0; i < nt; i++)
  {
    fprintf(fldfile, "%d %d %d ", m_meshPtr->faces[i][0], m_meshPtr->faces[i][1], m_meshPtr->faces[i][2]);
  }
  fprintf(fldfile, "}\n");
  fprintf(fldfile, "{TriLinearLgn<Point>  1 }\n}\n}{ConstantBasis<float>  1 }\n");
  fprintf(fldfile, "{STLVector 2 %d ", nt);
  for (size_t i = 0; i < nt; i++)
  {
    fprintf(fldfile, " 0");
  }

  fprintf(fldfile, "}\n}\n}");
  for (size_t i = 0; i < nv; i++)
  {
    fprintf(matfile, "%.12f\n", m_meshPtr->vertT[i]);
  }
  fclose(matfile);
  fclose(fldfile);
}

void meshFIM2d::writeVTK(std::vector< std::vector <float> > time_values)
{
  size_t nv = m_meshPtr->vertices.size();
  size_t nt = m_meshPtr->faces.size();
  for (size_t j = 0; j < time_values.size(); j++) {
    FILE* vtkfile;
    std::stringstream ss;
    ss << "result" << j << ".vtk";
    vtkfile = fopen(ss.str().c_str(), "w+");
    fprintf(vtkfile,
      "# vtk DataFile Version 3.0\nvtk output\nASCII\nDATASET UNSTRUCTURED_GRID\n");
    fprintf(vtkfile, "POINTS %d float\n", nv);
    for (size_t i = 0; i < nv; i++)
    {
      fprintf(vtkfile, "%.12f %.12f %.12f\n", m_meshPtr->vertices[i][0], 
        m_meshPtr->vertices[i][1], m_meshPtr->vertices[i][2]);
    }
    fprintf(vtkfile, "CELLS %d %d\n", nt, nt * 4);
    for (size_t i = 0; i < nt; i++)
    {
      fprintf(vtkfile, "3 %d %d %d\n", m_meshPtr->faces[i][0],
        m_meshPtr->faces[i][1], m_meshPtr->faces[i][2]);
    }

    fprintf(vtkfile, "CELL_TYPES %d\n", nt);
    for (size_t i = 0; i < nt; i++)
    {
      fprintf(vtkfile, "5\n");
    }
    fprintf(vtkfile,
      "POINT_DATA %d\nSCALARS traveltime float 1\nLOOKUP_TABLE default\n", nv);
    for (size_t i = 0; i < nv; i++)
    {
      fprintf(vtkfile, "%.12f\n", time_values[j][i]);
    }
    fclose(vtkfile);
  }
}

void meshFIM2d::updateT_single_stage_d(double timestep, int niter, 
  IdxVector_d& narrowband, int num_narrowband)
{
  size_t nn = m_meshPtr->vertices.size();
  int nblocks = num_narrowband;
  int nthreads = largest_ele_part;
  thrust::fill(vertT_out.begin(), vertT_out.end(), 0.0);
  int shared_size = sizeof (double)* 3 * largest_ele_part + 
    sizeof (short) *largest_vert_part*m_largest_num_inside_mem;

  cudaSafeCall((
    kernel_updateT_single_stage2d << <nblocks, nthreads, shared_size >> >(
    timestep,
    CAST(narrowband),
    largest_ele_part,
    largest_vert_part,
    full_num_ele,
    CAST(m_ele_after_permute_d),
    CAST(m_ele_offsets_d), 
    CAST(m_cadv_local_d),
    nn, CAST(m_vert_offsets_d),
    CAST(m_vert_after_permute_d), 
    CAST(m_vertT_after_permute_d),
    CAST(m_ele_local_coords_d),
    m_largest_num_inside_mem, 
    CAST(m_mem_locations), 
    CAST(m_mem_location_offsets),
    CAST(vertT_out))));
  nthreads = largest_vert_part;

  cudaSafeCall((CopyOutBack_levelset2d << <nblocks, nthreads >> >(
    CAST(narrowband),
    CAST(m_vert_offsets_d), 
    CAST(m_vertT_after_permute_d), 
    CAST(vertT_out))));
}

//Single stage update

void meshFIM2d::updateT_single_stage(double timestep, int nside, 
  int niter, std::vector<int>& narrowband)
{
  size_t nv = m_meshPtr->vertices.size();
  size_t nt = m_meshPtr->faces.size();
  std::vector<double> values(4);
  std::vector<double> up(nv, 0.0);
  std::vector<double> down(nv, 0.0);
  std::vector<vec3> node_grad_phi_up(nv, vec3(0.0, 0.0, 0.0));
  std::vector<double> node_grad_phi_down(nv, 0.0);
  std::vector<double> curv_up(nv, 0.0);


  for (size_t bandidx = 0; bandidx < narrowband.size(); bandidx++)
  {
    size_t tidx = narrowband[bandidx];
    vec3 sigma = m_meshPtr->normals[tidx];
    for (size_t j = 0; j < 4; j++)
    {
      values[j] = m_meshPtr->vertT[m_meshPtr->faces[tidx][j]];
    }
    //compute ni normals
    std::vector<vec3> nodes(4);
    nodes[0] = (vec3) m_meshPtr->vertices[m_meshPtr->faces[tidx][0]];
    nodes[1] = (vec3) m_meshPtr->vertices[m_meshPtr->faces[tidx][1]];
    nodes[2] = (vec3) m_meshPtr->vertices[m_meshPtr->faces[tidx][2]];
    nodes[3] = (vec3) m_meshPtr->vertices[m_meshPtr->faces[tidx][3]];
    vec3 v31 = nodes[1] - nodes[3];
    vec3 v32 = nodes[2] - nodes[3];
    vec3 v30 = nodes[0] - nodes[3];
    vec3 crossproduct = v31 CROSS v32;
    double dotproduct = crossproduct DOT v30;
    double volume = fabs(dotproduct) / 6.0;

    //compute inverse of 4 by 4 matrix
    double a11 = nodes[0][0], a12 = nodes[0][1], a13 = nodes[0][2], a14 = 1.0;
    double a21 = nodes[1][0], a22 = nodes[1][1], a23 = nodes[1][2], a24 = 1.0;
    double a31 = nodes[2][0], a32 = nodes[2][1], a33 = nodes[2][2], a34 = 1.0;
    double a41 = nodes[3][0], a42 = nodes[3][1], a43 = nodes[3][2], a44 = 1.0;

    double det =
      a11 * a22 * a33 * a44 + a11 * a23 * a34 * a42 + a11 * a24 * a32 * a43
      + a12 * a21 * a34 * a43 + a12 * a23 * a31 * a44 + a12 * a24 * a33 * a41
      + a13 * a21 * a32 * a44 + a13 * a22 * a34 * a41 + a13 * a24 * a31 * a42
      + a14 * a21 * a33 * a42 + a14 * a22 * a31 * a43 + a14 * a23 * a32 * a41
      - a11 * a22 * a34 * a43 - a11 * a23 * a32 * a44 - a11 * a24 * a33 * a42
      - a12 * a21 * a33 * a44 - a12 * a23 * a34 * a41 - a12 * a24 * a31 * a43
      - a13 * a21 * a34 * a42 - a13 * a22 * a31 * a44 - a13 * a24 * a32 * a41
      - a14 * a21 * a32 * a43 - a14 * a22 * a33 * a41 - a14 * a23 * a31 * a42;

    double b11 = a22 * a33 * a44 + a23 * a34 * a42 + a24 * a32 * a43 - a22 * a34 * a43 - a23 * a32 * a44 - a24 * a33 * a42;
    double b12 = a12 * a34 * a43 + a13 * a32 * a44 + a14 * a33 * a42 - a12 * a33 * a44 - a13 * a34 * a42 - a14 * a32 * a43;
    double b13 = a12 * a23 * a44 + a13 * a24 * a42 + a14 * a22 * a43 - a12 * a24 * a43 - a13 * a22 * a44 - a14 * a23 * a42;
    double b14 = a12 * a24 * a33 + a13 * a22 * a34 + a14 * a23 * a32 - a12 * a23 * a34 - a13 * a24 * a32 - a14 * a22 * a33;

    double b21 = a21 * a34 * a43 + a23 * a31 * a44 + a24 * a33 * a41 - a21 * a33 * a44 - a23 * a34 * a41 - a24 * a31 * a43;
    double b22 = a11 * a33 * a44 + a13 * a34 * a41 + a14 * a31 * a43 - a11 * a34 * a43 - a13 * a31 * a44 - a14 * a33 * a41;
    double b23 = a11 * a24 * a43 + a13 * a21 * a44 + a14 * a23 * a41 - a11 * a23 * a44 - a13 * a24 * a41 - a14 * a21 * a43;
    double b24 = a11 * a23 * a34 + a13 * a24 * a31 + a14 * a21 * a33 - a11 * a24 * a33 - a13 * a21 * a34 - a14 * a23 * a31;


    double b31 = a21 * a32 * a44 + a22 * a34 * a41 + a24 * a31 * a42 - a21 * a34 * a42 - a22 * a31 * a44 - a24 * a32 * a41;
    double b32 = a11 * a34 * a42 + a12 * a31 * a44 + a14 * a32 * a41 - a11 * a32 * a44 - a12 * a34 * a41 - a14 * a31 * a42;
    double b33 = a11 * a22 * a44 + a12 * a24 * a41 + a14 * a21 * a42 - a11 * a24 * a42 - a12 * a21 * a44 - a14 * a22 * a41;
    double b34 = a11 * a24 * a32 + a12 * a21 * a34 + a14 * a22 * a31 - a11 * a22 * a34 - a12 * a24 * a31 - a14 * a21 * a32;

    double b41 = a21 * a33 * a42 + a22 * a31 * a43 + a23 * a32 * a41 - a21 * a32 * a43 - a22 * a33 * a41 - a23 * a31 * a42;
    double b42 = a11 * a32 * a43 + a12 * a33 * a41 + a13 * a31 * a42 - a11 * a33 * a42 - a12 * a31 * a43 - a13 * a32 * a41;
    double b43 = a11 * a23 * a42 + a12 * a21 * a43 + a13 * a22 * a41 - a11 * a22 * a43 - a12 * a23 * a41 - a13 * a21 * a42;
    double b44 = a11 * a22 * a33 + a12 * a23 * a31 + a13 * a21 * a32 - a11 * a23 * a32 - a12 * a21 * a33 - a13 * a22 * a31;

    std::vector<vec4> Arows(4);
    Arows[0] = vec4(b11 / det, b12 / det, b13 / det, b14 / det);
    Arows[1] = vec4(b21 / det, b22 / det, b23 / det, b24 / det);
    Arows[2] = vec4(b31 / det, b32 / det, b33 / det, b34 / det);
    Arows[3] = vec4(b41 / det, b42 / det, b43 / det, b44 / det);

    std::vector<vec3> nablaN(4);
    for (int i = 0; i < 4; i++)
    {
      vec4 RHS(0.0, 0.0, 0.0, 0.0);
      RHS[i] = 1.0;
      nablaN[i][0] = Arows[0] DOT RHS;
      nablaN[i][1] = Arows[1] DOT RHS;
      nablaN[i][2] = Arows[2] DOT RHS;
    }

    //compuate grad of Phi
    vec3 nablaPhi(0.0, 0.0, 0.0);
    for (int i = 0; i < 4; i++)
    {
      nablaPhi[0] += nablaN[i][0] * values[i];
      nablaPhi[1] += nablaN[i][1] * values[i];
      nablaPhi[2] += nablaN[i][2] * values[i];
    }
    double abs_nabla_phi = len(nablaPhi);

    //compute K and Kplus and Kminus
    std::vector<double> Kplus(4);
    std::vector<double> Kminus(4);
    std::vector<double> K(4);
    double Hintegral = 0.0;
    double beta = 0;
    for (int i = 0; i < 4; i++)
    {
      K[i] = volume * (sigma DOT nablaN[i]); 
      Hintegral += K[i] * values[i];
      Kplus[i] = std::max(K[i], 0.0);
      Kminus[i] = std::min(K[i], 0.0);
      beta += Kminus[i];
    }
    beta = 1.0 / beta;

    if (fabs(Hintegral) > 1e-16)
    {
      std::vector<double> delta(4);
      for (int i = 0; i < 4; i++)
      {
        delta[i] = Kplus[i] * beta * (Kminus[0] * (values[i] - 
          values[0]) + Kminus[1] * (values[i] - values[1]) + Kminus[2] * 
          (values[i] - values[2]) + Kminus[3] * (values[i] - values[3]));
      }

      std::vector<double> alpha(4);
      for (int i = 0; i < 4; i++)
      {
        alpha[i] = delta[i] / Hintegral;
      }

      double theta = 0;
      for (int i = 0; i < 4; i++)
      {
        theta += std::max(0.0, alpha[i]);
      }

      std::vector<double> alphatuda(4);
      for (int i = 0; i < 4; i++)
      {
        alphatuda[i] = std::max(alpha[i], 0.0) / theta;
      }

      for (int i = 0; i < 4; i++)
      {
        up[m_meshPtr->faces[tidx][i]] += alphatuda[i] * Hintegral;
        down[m_meshPtr->faces[tidx][i]] += alphatuda[i] * volume;
        node_grad_phi_up[m_meshPtr->faces[tidx][i]] += nablaPhi * static_cast<float>(volume);
        node_grad_phi_down[m_meshPtr->faces[tidx][i]] += volume;
        curv_up[m_meshPtr->faces[tidx][i]] += volume * ((nablaN[i] DOT nablaN[i]) / abs_nabla_phi * values[i] +
            (nablaN[i] DOT nablaN[(i + 1) % 4]) / abs_nabla_phi * values[(i + 1) % 4] +
            (nablaN[i] DOT nablaN[(i + 2) % 4]) / abs_nabla_phi * values[(i + 2) % 4] +
            (nablaN[i] DOT nablaN[(i + 3) % 4]) / abs_nabla_phi * values[(i + 3) % 4]);
      }
    }
  }

  for (size_t vidx = 0; vidx < nv; vidx++)
  {
    double eikonal = up[vidx] / down[vidx];
    double curvature = curv_up[vidx] / node_grad_phi_down[vidx];
    double node_eikonal = len(node_grad_phi_up[vidx]) / node_grad_phi_down[vidx];
    if (fabs(down[vidx]) > 1e-16)
    {
      m_meshPtr->vertT[vidx] -= eikonal * timestep;
    }
  }
}

void meshFIM2d::GraphPartition_Square(int squareLength, int squareWidth, 
  int blockLength, int blockWidth, bool verbose)
{
  size_t nn = m_meshPtr->vertices.size();
  int numBlockLength = ceil((double) squareLength / blockLength);
  int numBlockWidth = ceil((double) squareWidth / blockWidth);
  int numBlock = numBlockLength * numBlockWidth;
  npart_h = IdxVector_h(nn);
  nparts = numBlock;

  int edgeCount = 0;
  for (size_t vIt = 0; vIt < nn; vIt++)
  {
    edgeCount += m_meshPtr->neighbors[vIt].size();
  }

  m_largest_num_inside_mem = 0;
  for (size_t i = 0; i < nn; i++)
  {
    if (m_meshPtr->adjacentfaces[i].size() > m_largest_num_inside_mem)
      m_largest_num_inside_mem = m_meshPtr->adjacentfaces[i].size();
  }

  if (verbose)
    printf("m_largest_num_inside_mem = %d\n", m_largest_num_inside_mem);

  //Allocating storage for array values of adjacency
  int* xadj = new int[nn + 1];
  int* adjncy = new int[edgeCount];

  // filling the arrays:
  xadj[0] = 0;
  int idx = 0;
  IdxVector_h neighbor_sizes(nn);
  // Populating the arrays:
  for (size_t i = 1; i < nn + 1; i++)
  {
    neighbor_sizes[i - 1] = m_meshPtr->neighbors[i - 1].size();
    xadj[i] = xadj[i - 1] + m_meshPtr->neighbors[i - 1].size();
    for (int j = 0; j < m_meshPtr->neighbors[i - 1].size(); j++)
    {
      adjncy[idx++] = m_meshPtr->neighbors[i - 1][j];
    }
  }

  m_neighbor_sizes_d = neighbor_sizes;

  for (size_t i = 0; i < squareWidth; i++)
    for (size_t j = 0; j < squareLength; j++)
    {
      int index = i * squareLength + j;
      int i2 = i;
      int j2 = j;
      npart_h[index] = (i2 / blockWidth) * numBlockLength + (j2 / blockLength);
    }

  m_xadj_d = IdxVector_d(&xadj[0], &xadj[nn + 1]);
  m_adjncy_d = IdxVector_d(&adjncy[0], &adjncy[edgeCount]);

  IdxVector_h part_sizes(nparts, 0);
  for (size_t i = 0; i < nn; i++)
  {
    part_sizes[npart_h[i]]++;
  }
  int min_part_size = thrust::reduce(part_sizes.begin(), part_sizes.end(), 100000000, thrust::minimum<int>());
  largest_vert_part = thrust::reduce(part_sizes.begin(), part_sizes.end(), -1, thrust::maximum<int>());

  if (verbose)
    printf("Largest vertex partition size is: %d\n", largest_vert_part);
  if (min_part_size == 0) printf("Min partition size is 0!!\n");
  delete [] xadj;
  delete [] adjncy;
}

void meshFIM2d::Partition_METIS(int metissize, bool verbose)
{
  int options[10], pnumflag = 0, wgtflag = 0;
  options[0] = 0;
  int edgecut;
  int nn = m_meshPtr->vertices.size();
  npart_h = IdxVector_h(nn);
  nparts = nn / metissize;
  if (nparts < 2)
    nparts = 2;

  // Counting up edges for adjacency:
  int edgeCount = 0;
  for (int vIt = 0; vIt < nn; vIt++)
  {
    edgeCount += m_meshPtr->neighbors[vIt].size();
  }

  m_largest_num_inside_mem = 0;
  //for(int bidx = 0; bidx < nparts; bidx++)
  for (int i = 0; i < nn; i++)
  {
    if (m_meshPtr->adjacentfaces[i].size() > m_largest_num_inside_mem)
      m_largest_num_inside_mem = m_meshPtr->adjacentfaces[i].size();
  }

  if (verbose)
    printf("m_largest_num_inside_mem = %d\n", m_largest_num_inside_mem);

  //Allocating storage for array values of adjacency
  int* xadj = new int[nn + 1];
  int* adjncy = new int[edgeCount];

  // filling the arrays:
  xadj[0] = 0;
  int idx = 0;
  IdxVector_h neighbor_sizes(nn);
  // Populating the arrays:
  for (int i = 1; i < nn + 1; i++)
  {
    neighbor_sizes[i - 1] = m_meshPtr->neighbors[i - 1].size();
    xadj[i] = xadj[i - 1] + m_meshPtr->neighbors[i - 1].size();
    for (int j = 0; j < m_meshPtr->neighbors[i - 1].size(); j++)
    {
      adjncy[idx++] = m_meshPtr->neighbors[i - 1][j];
    }
  }

  m_neighbor_sizes_d = neighbor_sizes;
  int* npart_h_ptr = thrust::raw_pointer_cast(&npart_h[0]);

  METIS_PartGraphKway(&nn, xadj, adjncy, NULL, NULL, &wgtflag, 
    &pnumflag, &nparts, options, &edgecut, npart_h_ptr);

  m_xadj_d = IdxVector_d(&xadj[0], &xadj[nn + 1]);
  m_adjncy_d = IdxVector_d(&adjncy[0], &adjncy[edgeCount]);

  IdxVector_h part_sizes(nparts, 0);
  for (int i = 0; i < nn; i++)
  {
    part_sizes[npart_h[i]]++;
  }
  int min_part_size = thrust::reduce(part_sizes.begin(), 
    part_sizes.end(), 100000000, thrust::minimum<int>());
  largest_vert_part = thrust::reduce(part_sizes.begin(), 
    part_sizes.end(), -1, thrust::maximum<int>());

  if (verbose)
    printf("Largest vertex partition size is: %d\n", largest_vert_part);
  if (min_part_size == 0) printf("Min partition size is 0!!\n");
  delete [] xadj;
  delete [] adjncy;
}

void meshFIM2d::InitPatches(bool verbose)
{
  int ne = m_meshPtr->faces.size();
  int nn = m_meshPtr->vertices.size();
  ele_d = IdxVector_d(3 * ne);
  ele_h = IdxVector_h(3 * ne);
  vert_d = Vector_d(3 * nn);
  m_vert_after_permute_d = Vector_d(3 * nn);
  Vector_h vert_h(3 * nn);
  for (int eidx = 0; eidx < ne; eidx++)
  {
    for (int i = 0; i < 3; i++)
      ele_h[i * ne + eidx] = m_meshPtr->faces[eidx][i]; //interleaved storage
  }
  for (int vidx = 0; vidx < nn; vidx++)
  {
    for (int i = 0; i < 3; i++)
      vert_h[i * nn + vidx] = m_meshPtr->vertices[vidx][i]; //interleaved storage
  }
  ele_d = ele_h;
  vert_d = vert_h;
  m_npart_d = IdxVector_d(npart_h.begin(), npart_h.end());
  m_part_label_d = IdxVector_d(m_npart_d.begin(), m_npart_d.end());
  int nthreads = 256;
  int nblocks = min((int) ceil((double) ne / nthreads), 65535);
  IdxVector_d ele_label_d(ne);
  cudaSafeCall((kernel_compute_ele_npart2d << <nblocks, nthreads >> >(
    ne, thrust::raw_pointer_cast(&m_npart_d[0]),
    thrust::raw_pointer_cast(&ele_d[0]), 
    thrust::raw_pointer_cast(&ele_label_d[0]))));

  full_num_ele = thrust::reduce(ele_label_d.begin(), ele_label_d.end());

  if (verbose)
    printf("full_num_ele = %d\n", full_num_ele);
  IdxVector_d ele_offsets_d(ne + 1);
  ele_offsets_d[0] = 0;
  thrust::inclusive_scan(ele_label_d.begin(), ele_label_d.end(), ele_offsets_d.begin() + 1);
  IdxVector_d ele_full_label(full_num_ele);
  ele_permute = IdxVector_d(full_num_ele);
  cudaSafeCall((kernel_fill_ele_label2d << <nblocks, nthreads >> >(ne,
    thrust::raw_pointer_cast(&ele_permute[0]),
    thrust::raw_pointer_cast(&ele_offsets_d[0]),
    thrust::raw_pointer_cast(&m_npart_d[0]),
    thrust::raw_pointer_cast(&ele_d[0]),
    thrust::raw_pointer_cast(&ele_full_label[0]))));
  thrust::sort_by_key(ele_full_label.begin(), ele_full_label.end(),
    ele_permute.begin());
  m_ele_offsets_d = IdxVector_d(nparts + 1);
  IdxVector_d ones(full_num_ele, 1);
  IdxVector_d tmp(full_num_ele);
  IdxVector_d reduce_output(full_num_ele);
  thrust::reduce_by_key(ele_full_label.begin(), ele_full_label.end(),
    ones.begin(), tmp.begin(), reduce_output.begin());
  largest_ele_part = thrust::reduce(reduce_output.begin(), 
    reduce_output.begin() + nparts, -1, thrust::maximum<int>());
  if (verbose)
    printf("Largest element partition size is: %d\n", largest_ele_part);
  if (largest_ele_part > 1024)
  {
    printf("Error: largest_ele_part > 1024 !!\n");
    exit(0);
  }
  m_ele_offsets_d[0] = 0;
  thrust::inclusive_scan(reduce_output.begin(),
    reduce_output.begin() + nparts, m_ele_offsets_d.begin() + 1);
}

void meshFIM2d::InitPatches2()
{
  int ne = m_meshPtr->faces.size();
  int nn = m_meshPtr->vertices.size();
  //compute invert vert permute
  IdxVector_d vert_permute(nn, 0);
  IdxVector_d vert_ipermute(nn, 0);
  int nthreads = 256;
  int nblocks = min((int) ceil((double) nn / nthreads), 65535);
  cudaSafeCall((kernel_fill_sequence2d << <nblocks, nthreads >> >(nn, CAST(vert_permute))));
  thrust::sort_by_key(m_part_label_d.begin(), m_part_label_d.end(), vert_permute.begin());
  nblocks = min((int) ceil((double) nn / nthreads), 65535);

  cudaSafeCall((kernel_compute_vert_ipermute2d << <nblocks, nthreads >> >(nn,
    thrust::raw_pointer_cast(&vert_permute[0]),
    thrust::raw_pointer_cast(&vert_ipermute[0]))));
  m_vert_permute_d = IdxVector_d(vert_permute);
  m_vert_offsets_d = IdxVector_d(nparts + 1);
  cusp::detail::indices_to_offsets(m_part_label_d, m_vert_offsets_d);
  //permute the vert and ele values and Rinscribe
  m_ele_after_permute_d = IdxVector_d(3 * full_num_ele);
  m_vertT_after_permute_d = Vector_d(nn);
  nblocks = min((int)ceil((double)full_num_ele / nthreads), 65535);
  cudaSafeCall((kernel_ele_and_vert2d << <nblocks, nthreads >> >(full_num_ele, ne,
    thrust::raw_pointer_cast(&ele_d[0]),
    thrust::raw_pointer_cast(&m_ele_after_permute_d[0]),
    thrust::raw_pointer_cast(&ele_permute[0]),
    nn, thrust::raw_pointer_cast(&vert_d[0]),
    thrust::raw_pointer_cast(&m_vert_after_permute_d[0]),
    thrust::raw_pointer_cast(&m_vertT_d[0]),
    thrust::raw_pointer_cast(&m_vertT_after_permute_d[0]),
    CAST(m_Rinscribe_before_permute_d),
    CAST(m_Rinscribe_d),
    CAST(vert_permute),
    thrust::raw_pointer_cast(&vert_ipermute[0]))));
  //compute the local coords for each element
  m_ele_local_coords_d = Vector_d(3 * full_num_ele);
  m_cadv_local_d = Vector_d(2 * full_num_ele);
  nthreads = 256;
  nblocks = min((int)ceil((float)full_num_ele / nthreads), 65535);
  cudaSafeCall((kernel_compute_local_coords2d << <nblocks, nthreads >> >(
    full_num_ele, nn,
    thrust::raw_pointer_cast(&m_ele_after_permute_d[0]),
    thrust::raw_pointer_cast(&m_ele_offsets_d[0]),
    thrust::raw_pointer_cast(&m_vert_after_permute_d[0]),
    thrust::raw_pointer_cast(&m_ele_local_coords_d[0]),
    CAST(m_cadv_global_d),
    CAST(m_cadv_local_d))));
  //Generate redution list

  m_mem_locations = IdxVector_d(3 * full_num_ele);
  IdxVector_d tmp2 = m_ele_after_permute_d;
  thrust::sequence(m_mem_locations.begin(), m_mem_locations.end(), 0);
  thrust::sort_by_key(tmp2.begin(), tmp2.end(), m_mem_locations.begin());
  m_mem_location_offsets = IdxVector_d(nn + 1);
  cusp::detail::indices_to_offsets(tmp2, m_mem_location_offsets);
}

void meshFIM2d::GenerateBlockNeighbors()
{

  //Generate block neighbors
  // Declaring temporary vectors:
  IdxVector_d adjacencyBlockLabel, blockMappedAdjacency;
  adjacencyBlockLabel = IdxVector_d(m_adjncy_d.size(), 0);
  blockMappedAdjacency = IdxVector_d(m_adjncy_d.size(), 0);

  mapAdjacencyToBlock(m_xadj_d, m_adjncy_d, adjacencyBlockLabel, blockMappedAdjacency, m_npart_d);

  // Zip up the block label and block mapped vectors and sort:
  thrust::sort(thrust::make_zip_iterator(thrust::make_tuple(
    adjacencyBlockLabel.begin(), blockMappedAdjacency.begin())),
      thrust::make_zip_iterator(
      thrust::make_tuple(adjacencyBlockLabel.end(),
      blockMappedAdjacency.end())));

  // Remove Duplicates and resize:
  int newSize = thrust::unique(thrust::make_zip_iterator(
    thrust::make_tuple(adjacencyBlockLabel.begin(),
    blockMappedAdjacency.begin())),
      thrust::make_zip_iterator(thrust::make_tuple(
      adjacencyBlockLabel.end(), blockMappedAdjacency.end()))) -
    thrust::make_zip_iterator(thrust::make_tuple(
    adjacencyBlockLabel.begin(), blockMappedAdjacency.begin()));

  adjacencyBlockLabel.resize(newSize);
  blockMappedAdjacency.resize(newSize);
  getPartIndicesNegStart(adjacencyBlockLabel, m_block_xadj_d);
  m_block_adjncy_d.resize(blockMappedAdjacency.size() - 1);
  thrust::copy(blockMappedAdjacency.begin() + 1, 
    blockMappedAdjacency.end(), m_block_adjncy_d.begin());

}

void meshFIM2d::compute_deltaT(int num_narrowband, bool verbose)
{
  int nn = m_meshPtr->vertices.size();
  int ne = m_meshPtr->faces.size();
  int nnb = num_narrowband;

  m_timestep = LARGENUM;
  int nblocks = nnb;
  int nthreads = largest_ele_part;

  if (nthreads <= 32)
  {
    cudaSafeCall((kernel_compute_timestep2d < 32 > << <nblocks, 32 >> >(
      full_num_ele, CAST(m_narrowband_d), CAST(m_ele_offsets_d),
      CAST(m_Rinscribe_d), CAST(m_cadv_local_d), CAST(m_ceik_global_d), 
      CAST(m_ccurv_global_d), CAST(timestep_per_block), CAST(Rin_per_block))));
  }
  else if (nthreads <= 64)
  {
    cudaSafeCall((kernel_compute_timestep2d < 64 > << <nblocks, 64 >> >(
      full_num_ele, CAST(m_narrowband_d), CAST(m_ele_offsets_d), 
      CAST(m_Rinscribe_d), CAST(m_cadv_local_d), CAST(m_ceik_global_d), 
      CAST(m_ccurv_global_d), CAST(timestep_per_block), CAST(Rin_per_block))));
  }
  else if (nthreads <= 128)
  {
    cudaSafeCall((kernel_compute_timestep2d < 128 > << <nblocks, 128 >> >(
      full_num_ele, CAST(m_narrowband_d), CAST(m_ele_offsets_d),
      CAST(m_Rinscribe_d), CAST(m_cadv_local_d), CAST(m_ceik_global_d),
      CAST(m_ccurv_global_d), CAST(timestep_per_block), CAST(Rin_per_block))));
  }
  else if (nthreads <= 256)
  {
    cudaSafeCall((kernel_compute_timestep2d < 256 > << <nblocks, 256 >> >(
      full_num_ele, CAST(m_narrowband_d), CAST(m_ele_offsets_d), 
      CAST(m_Rinscribe_d), CAST(m_cadv_local_d), CAST(m_ceik_global_d), 
      CAST(m_ccurv_global_d), CAST(timestep_per_block), CAST(Rin_per_block))));
  }
  else if (nthreads <= 512)
  {
    cudaSafeCall((kernel_compute_timestep2d < 512 > << <nblocks, 512 >> >(
      full_num_ele, CAST(m_narrowband_d), CAST(m_ele_offsets_d),
      CAST(m_Rinscribe_d), CAST(m_cadv_local_d), CAST(m_ceik_global_d), 
      CAST(m_ccurv_global_d), CAST(timestep_per_block), CAST(Rin_per_block))));
  }
  else if (nthreads <= 1024)
  {
    cudaSafeCall((kernel_compute_timestep2d < 1024 > << <nblocks, 1024 >> >(
         full_num_ele, CAST(m_narrowband_d), CAST(m_ele_offsets_d),
      CAST(m_Rinscribe_d), CAST(m_cadv_local_d), CAST(m_ceik_global_d), 
      CAST(m_ccurv_global_d), CAST(timestep_per_block), CAST(Rin_per_block))));
  }
  else
  {
    printf("Error: nthreads greater than 256!!!\n");
  }

  m_timestep = thrust::reduce(timestep_per_block.begin(), 
    timestep_per_block.begin() + nnb, (double) LARGENUM,
    thrust::minimum<double > ());
  m_maxRin = -1.0;
  m_maxRin = thrust::reduce(Rin_per_block.begin(),
    Rin_per_block.begin() + nnb, (double) - 1.0, 
    thrust::maximum<double > ());

}

std::vector< std::vector< float > > meshFIM2d::GenerateData(
    const char* filename, int nsteps,
    double timestep, int inside_niter,
    int nside, int block_size, double bandwidth,
    int part_type, int metis_size, bool verbose)
{

  if (verbose)
    printf("Starting meshFIM2d::GenerateData\n");
  int nv = m_meshPtr->vertices.size();
  int nt = m_meshPtr->faces.size();

  int squareLength = nside;
  int squareWidth = nside;
  int squareBlockLength = block_size;
  int squareBlockWidth = block_size;
  clock_t starttime, endtime;
  double duration;

  if (part_type == 1)
    GraphPartition_Square(squareLength, squareWidth, 
    squareBlockLength, squareBlockWidth,verbose);
  else //partition with METIS
  {
    Partition_METIS(metis_size, verbose);
  }

  if (m_meshPtr->vertT.size() == 0)
    m_meshPtr->vertT.resize(nv);

  Vector_h h_vertT(nv);
  for (int i = 0; i < nv; i++)
  {
    h_vertT[i] = m_meshPtr->vertT[i];
  }
  m_vertT_d = h_vertT;
  m_Rinscribe_before_permute_d = Vector_d(m_meshPtr->radiusInscribe);
  largest_Rin = thrust::reduce(m_Rinscribe_before_permute_d.begin(),
      m_Rinscribe_before_permute_d.end(), -1, thrust::maximum<double > ());

  starttime = clock();
  
  //Init patches
  InitPatches(verbose);
  Vector_h cadv_h(3 * full_num_ele, 0);
  Vector_h ceik_h(full_num_ele);
  Vector_h ccurv_h(full_num_ele);
  IdxVector_h ele_permute_h = IdxVector_h(ele_permute);
  for (int i = 0; i < full_num_ele; i++) {
    size_t triIdx = static_cast<size_t>(ele_permute_h[i]);
    ceik_h[i] = 10.0f;
    ccurv_h[i] = 0.0f;
    cadv_h[0 * full_num_ele + i] = m_meshPtr->normals[triIdx][0];
    cadv_h[1 * full_num_ele + i] = m_meshPtr->normals[triIdx][1];
    cadv_h[2 * full_num_ele + i] = m_meshPtr->normals[triIdx][2];
  }
  m_cadv_global_d = Vector_d(cadv_h);
  m_ceik_global_d = Vector_d(ceik_h);
  m_ccurv_global_d = Vector_d(ccurv_h);
  m_Rinscribe_d = Vector_d(full_num_ele);
  
  InitPatches2();
  GenerateBlockNeighbors();
  hipDeviceSynchronize();
  
  if (verbose) 
    printf("After  preprocessing\n");
  endtime = clock();
  duration = (double)(endtime - starttime) / CLOCKS_PER_SEC;
  if (verbose)
    printf("pre processing time : %.10lf s\n", duration);
  
  //Inite redistance
  m_redist = new redistance(m_meshPtr, nparts,
    m_block_xadj_d, m_block_adjncy_d);

  //////////////////////////update values//////////////////////////
  m_narrowband_d = IdxVector_d(nparts);
  timestep_per_block = Vector_d(nparts);
  Rin_per_block = Vector_d(nparts, -1);
  int num_narrowband = 0;

  starttime = clock();
  std::vector< std::vector< float > > data;
  data.push_back(m_meshPtr->vertT);
  for (int stepcount = 0; stepcount < nsteps; stepcount++)
  {
    m_redist->FindSeedPoint(
      m_narrowband_d, 
      num_narrowband,
      m_meshPtr, 
      m_vertT_after_permute_d,
      nparts,
      largest_vert_part, 
      largest_ele_part, 
      m_largest_num_inside_mem, 
      full_num_ele,
      m_vert_after_permute_d,
      m_vert_offsets_d, 
      m_ele_after_permute_d, 
      m_ele_offsets_d,
      m_ele_local_coords_d, 
      m_mem_location_offsets,
      m_mem_locations,
      m_part_label_d,
      m_block_xadj_d, 
      m_block_adjncy_d);

    m_redist->ReInitTsign(
      m_meshPtr, 
      m_vertT_after_permute_d,
      nparts, 
      largest_vert_part, 
      largest_ele_part, 
      m_largest_num_inside_mem,
      full_num_ele,
      m_vert_after_permute_d, 
      m_vert_offsets_d, 
      m_ele_after_permute_d, 
      m_ele_offsets_d, 
      m_ele_local_coords_d, 
      m_mem_location_offsets, 
      m_mem_locations,
      m_part_label_d, 
      m_block_xadj_d, 
      m_block_adjncy_d);

    m_redist->GenerateData(
      m_narrowband_d, 
      num_narrowband, 
      bandwidth, 
      stepcount, 
      m_meshPtr, 
      m_vertT_after_permute_d, 
      nparts, 
      largest_vert_part, 
      largest_ele_part, 
      m_largest_num_inside_mem, 
      full_num_ele,
      m_vert_after_permute_d,
      m_vert_offsets_d,
      m_ele_after_permute_d, 
      m_ele_offsets_d, 
      m_ele_local_coords_d, 
      m_mem_location_offsets, 
      m_mem_locations,
      m_part_label_d, 
      m_block_xadj_d, 
      m_block_adjncy_d, 
      verbose);

    if (num_narrowband == 0) {
      std::cout << "NOTE: Ending at timestep " << stepcount <<
        " due to zero narrow band." << std::endl;
      break;
    }
	
    compute_deltaT(num_narrowband, verbose);
    for (int niter = 0; niter < inside_niter; niter++)
      updateT_single_stage_d(timestep, stepcount, m_narrowband_d, num_narrowband);
    //////////////////////////done updating/////////////////////////////////////////////////
	
	// convert vertT back to non-permuted order for use
    int nthreads = 256;
    int nblocks = min((int)ceil((double)nv / nthreads), 655535);
    cudaSafeCall((
      kernel_compute_vertT_before_permute2d << <nblocks, nthreads >> >(
      nv, 
      CAST(m_vert_permute_d),
      CAST(m_vertT_after_permute_d), 
      CAST(tmp_vertT_before_permute_d))));

    Vector_h vertT_before_permute_h = tmp_vertT_before_permute_d;

    for (int i = 0; i < nv; i++)
    {
      m_meshPtr->vertT[i] = vertT_before_permute_h[i];
    }
    data.push_back(m_meshPtr->vertT);
  }

  hipDeviceSynchronize();
  endtime = clock();
  duration = (double) (endtime - starttime) / CLOCKS_PER_SEC;
  if (verbose)
    printf("Processing time : %.10lf s\n", duration);
  return data;
}

void meshFIM2d::getPartIndicesNegStart(IdxVector_d& sortedPartition, IdxVector_d& partIndices)
{
  // Sizing the array:
  int maxPart = sortedPartition[sortedPartition.size() - 1];
  partIndices.resize(maxPart + 2, 0);

  // Figuring out block sizes for kernel call:
  int size = sortedPartition.size();
  int blockSize = 256;
  int nBlocks = size / blockSize + (size % blockSize == 0 ? 0 : 1);

  // Getting pointers
  int *sortedPartition_d = thrust::raw_pointer_cast(&sortedPartition[0]);
  int *partIndices_d = thrust::raw_pointer_cast(&partIndices[0]);

  // Calling kernel to find indices for each part:
  findPartIndicesNegStartKernel2d << < nBlocks, blockSize >> > (size, sortedPartition_d, partIndices_d);
  partIndices[partIndices.size() - 1] = size - 1;
}

void meshFIM2d::mapAdjacencyToBlock(IdxVector_d &adjIndexes, IdxVector_d &adjacency,
  IdxVector_d &adjacencyBlockLabel, IdxVector_d &blockMappedAdjacency, IdxVector_d &fineAggregate)
{
  int size = adjIndexes.size() - 1;
  // Get pointers:adjacencyIn
  int *adjIndexes_d = thrust::raw_pointer_cast(&adjIndexes[0]);
  int *adjacency_d = thrust::raw_pointer_cast(&adjacency[0]);
  int *adjacencyBlockLabel_d = thrust::raw_pointer_cast(&adjacencyBlockLabel[0]);
  int *blockMappedAdjacency_d = thrust::raw_pointer_cast(&blockMappedAdjacency[0]);
  int *fineAggregate_d = thrust::raw_pointer_cast(&fineAggregate[0]);

  // Figuring out block sizes for kernel call:
  int blockSize = 256;
  int nBlocks = size / blockSize + (size % blockSize == 0 ? 0 : 1);

  // Calling kernel:
  mapAdjacencyToBlockKernel2d << < nBlocks, blockSize >> > (size, adjIndexes_d, adjacency_d, 
    adjacencyBlockLabel_d, blockMappedAdjacency_d, fineAggregate_d);
}

